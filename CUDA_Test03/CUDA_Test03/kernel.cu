#include "hip/hip_runtime.h"

#include <stdio.h>

// This will run on the GPU.
__global__ void add(int *a, int *b, int *c) {
	*c = *a + *b;
}

#define N 512

// This will run on the CPU.
int main(void) {
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);

	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	// Alloc space for host copies of a, b, c and setup input values
	//a = (int *)malloc(size); random_ints(a, N);
	//b = (int *)malloc(size); random_ints(b, N);
	//c = (int *)malloc(size);

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU with N blocks
	add<<<N,1>>>(d_a, d_b, d_c);

	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	// Cleanup
	//free(a);
	//free(b);
	//free(c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
