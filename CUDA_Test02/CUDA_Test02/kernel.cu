#include "hip/hip_runtime.h"

#include <stdio.h>

// device = GPU
// host = CPU

// This will run on the device (GPU).
__global__ void add(int *a, int *b, int *c) {
	*c = *a + *b;
}

int main(void) {
	// host copies of a, b, c
	int a, b, c;

	// device copies of a, b, c
	int *d_a, *d_b, *d_c;

	int size = sizeof(int);

	// Allocate space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	
	// Setup input values
	a = 4;
	b = 6;
	
	// Copy inputs to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU
	add<<<1,1>>>(d_a, d_b, d_c);
	
	// Copy result back to host
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	
	// Cleanup
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	printf("%d + %d = %d", a, b, c);
	
	return 0;
}
